// %%writefile ass4gpu.cu
#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <chrono>
#include <vector>
using namespace std;

__device__ bool isPointOnLine(int xc, int yc, int x1, int y1, int x2, int y2) {
    
    // Check if the line is vertical
    if (x1 == x2) {
        return xc == x1;  // Tank lies on the line if x-coordinate matches
    }
    else if (y1 == y2) {
        return yc == y1;  // Tank lies on the line if y-coordinate matches
    }
    else {
        // Calculate slopes using type casting to ensure floating-point division
        int slope1 = (yc - y1) * (x2 - x1);
        int slope2 = (y2 - y1) * (xc - x1);

        // Compare slopes to check if they are equal
        return slope1 == slope2;
    }
}

__global__ void tankInteractionKernel(int *x,int *y,int *scoring,int *hp, int T,int currentRound,int* hd_hitScoreCount){
    int tank_id = blockIdx.x;
    int tank_id_within = threadIdx.x;
    
    int target_id = (tank_id + currentRound) % T;  // Choose the next tank as the target
    int xc = x[threadIdx.x];
    int yc = y[threadIdx.x];
    __syncthreads();

    // Load tank coordinates and direction into shared memory
    __shared__ int x1, y1, x2, y2, dir;
    if (threadIdx.x == 0) {
        x1 = x[tank_id];
        y1 = y[tank_id];
        x2 = x[target_id];
        y2 = y[target_id];
        // Determine direction
        if (x1 <= x2 && y1 <= y2) dir = 1;
        else if (x1 > x2 && y1 <= y2) dir = 2;
        else if (x1 > x2 && y1 > y2) dir = 3;
        else if (x1 <= x2 && y1 > y2) dir = 4;
    }
    __syncthreads(); 

    bool result = isPointOnLine(xc, yc, x1, y1, x2, y2);
    
    __syncthreads();

    __shared__ int it, temp, old, lockvar;
    if (threadIdx.x == 0) {
        it = -1;
        temp = 1e9;
        old = 10;
        lockvar = 0;
    }
    
    __syncthreads();  // Ensure all threads have initialized shared memory
    


    do {
    old = atomicCAS(&lockvar, 0,1);
        if(old == 0){
            if (result && hp[tank_id] > 0 && tank_id != tank_id_within && dir == 1) {
                if (hp[tank_id_within] > 0 && x1 <= x[tank_id_within] && y1 <= y[tank_id_within]) {
                    if ((abs(x1 - x[tank_id_within]) + abs(y1 - y[tank_id_within])) < temp) {
                        temp = (abs(x1 - x[tank_id_within]) + abs(y1 - y[tank_id_within]));
                        it = tank_id_within;
                    }
                }
            }
            if (result && hp[tank_id] > 0 && tank_id != tank_id_within && dir == 2) {
                if (hp[tank_id_within] > 0 && x1 > x[tank_id_within] && y1 <= y[tank_id_within]) {
                    if ((abs(x1 - x[tank_id_within]) + abs(y1 - y[tank_id_within])) < temp) {
                        temp = (abs(x1 - x[tank_id_within]) + abs(y1 - y[tank_id_within]));
                        it = tank_id_within;
                    }
                }
            }
            if (result && hp[tank_id] > 0 && tank_id != tank_id_within && dir == 3) {
                if (hp[tank_id_within] > 0 && x1 > x[tank_id_within] && y1 > y[tank_id_within]) {
                    if ((abs(x1 - x[tank_id_within]) + abs(y1 - y[tank_id_within])) < temp) {
                        temp = (abs(x1 - x[tank_id_within]) + abs(y1 - y[tank_id_within]));
                        it = tank_id_within;
                    }
                }
            }
            if (result && hp[tank_id] > 0 && tank_id != tank_id_within && dir == 4) {
                if (hp[tank_id_within] > 0 && x1 <= x[tank_id_within] && y1 > y[tank_id_within]) {
                    if ((abs(x1 - x[tank_id_within]) + abs(y1 - y[tank_id_within])) < temp) {
                        temp = (abs(x1 - x[tank_id_within]) + abs(y1 - y[tank_id_within]));
                        it = tank_id_within;
                    }
                }
            }
            lockvar = 0;
        }
    } while(old!=0);
    __syncthreads();
    if (it != -1) {
        
        atomicAdd(&hd_hitScoreCount[it],1);
        atomicAdd(&hd_hitScoreCount[T+tank_id],1);
    }
    __syncthreads();

}



int main(int argc, char **argv) {
    // Variable declarations
    int M, N, T, H, *xcoord, *ycoord, *score;

    FILE *inputfilepointer;

    // File Opening for read
    char *inputfilename = argv[1];
    inputfilepointer = fopen(inputfilename, "r");

    if (inputfilepointer == NULL) {
        printf("input.txt file failed to open.");
        return 0;
    }

    fscanf(inputfilepointer, "%d", &M);
    fscanf(inputfilepointer, "%d", &N);
    fscanf(inputfilepointer, "%d", &T); // T is number of Tanks
    fscanf(inputfilepointer, "%d", &H); // H is the starting Health point of each Tank

    // Allocate memory on CPU
    xcoord = (int *)malloc(T * sizeof(int));  // X coordinate of each tank
    ycoord = (int *)malloc(T * sizeof(int));  // Y coordinate of each tank
    score = (int *)malloc(T * sizeof(int));  // Score of each tank (ensure that at the end you have copied back the score calculations on the GPU back to this allocation)

    // Get the Input of Tank coordinates
    for (int i = 0; i < T; i++) {
        fscanf(inputfilepointer, "%d", &xcoord[i]);
        fscanf(inputfilepointer, "%d", &ycoord[i]);
    }

    auto start = chrono::high_resolution_clock::now();

    //*********************************
    // Your Code begins here (Do not change anything in main() above this comment)
    //********************************

    
    int *x;   // X coordinate of each tank
    int *y;    // Y coordinate of each tank
    int *hp;   // Health Points of each tank
    int *scoring; // Score of each tank
    
    hipHostAlloc(&x, T*sizeof(int), 0);
    hipHostAlloc(&y, T*sizeof(int), 0);
    hipHostAlloc(&hp, T*sizeof(int), 0);
    hipHostAlloc(&scoring, T*sizeof(int), 0);
    
    
    
    for (int i = 0; i < T; ++i) {
        x[i] = xcoord[i];
        y[i] = ycoord[i];
        scoring[i] = score[i];
        hp[i] = H;
    }

    

    int tanksLeft = T;
    int currentRound = 1;
    while (tanksLeft > 1) {  // Continue rounds until only one or zero tanks left
        if (currentRound % T == 0) {
            currentRound++;
            continue;
        }
        int *hd_hitScoreCount;
        hipHostAlloc(&hd_hitScoreCount, 2*T*sizeof(int), 0);
        
        tankInteractionKernel<<<T,T>>>(x,y,scoring,hp, T, currentRound, hd_hitScoreCount);
        hipDeviceSynchronize();
        for(int i = 0; i < T; ++i) {
            printf("score %d , health %d \n", scoring[i], hp[i]);
            hp[i] -= hd_hitScoreCount[i];
            scoring[i] += hd_hitScoreCount[T+i];
            if (hp[i] <= 0 && hd_hitScoreCount[i]) tanksLeft--;
            
        }
        printf("score %d , health %d \n", scoring[T-1], hp[T-1]);

        currentRound++;
        hipHostFree(hd_hitScoreCount);

    }
    printf("checking code execuete properly or not");


    
    

    //*********************************
    // Your Code ends here (Do not change anything in main() below this comment)
    //********************************

    auto end = chrono::high_resolution_clock::now();

    chrono::duration<double, std::micro> timeTaken = end - start;

    printf("Execution time : %f\n", timeTaken.count());

    // Output
    char *outputfilename = argv[2];
    char *exectimefilename = argv[3];
    FILE *outputfilepointer;
    outputfilepointer = fopen(outputfilename, "w");

    for (int i = 0; i < T; i++) {
        fprintf(outputfilepointer, "%d\n", scoring[i]);
    }
    fclose(inputfilepointer);
    fclose(outputfilepointer);

    outputfilepointer = fopen(exectimefilename, "w");
    fprintf(outputfilepointer, "%f", timeTaken.count());
    fclose(outputfilepointer);

    free(xcoord);
    free(ycoord);
    free(score);
    hipHostFree(x);
    hipHostFree(y);
    hipHostFree(hp);
    hipHostFree(scoring);
    hipDeviceSynchronize();
    return 0;
}
