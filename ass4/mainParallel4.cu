// %%writefile ass4gpu1.cu
#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <chrono>
#include <vector>
using namespace std;
__device__ __host__ int calcDist(int x1, int y1, int x2, int y2){
    int dis = ((x2-x1)*(x2-x1)) + ((y2-y1)*(y2-y1));
    return dis; 
}
__device__ __host__ int calcDir(int x1, int y1, int x2, int y2){
    if (x1 <= x2 && y1 <= y2) return 1;
    else if (x1 > x2 && y1 <= y2) return 2;
    else if (x1 > x2 && y1 > y2) return 3;
    return 4;
}
__device__ __host__ bool isPointOnLine(int xc, int yc, int x1, int y1, int x2, int y2) {
    
    // Check if the line is vertical
    if (x1 == x2) {
        return xc == x1;  // Tank lies on the line if x-coordinate matches
    }
    else if (y1 == y2) {
        return yc == y1;  // Tank lies on the line if y-coordinate matches
    }
    else {
        // Calculate slopes using type casting to ensure floating-point division
        int slope1 = (yc - y1) * (x2 - x1);
        int slope2 = (y2 - y1) * (xc - x1);

        // Compare slopes to check if they are equal
        return slope1 == slope2;
    }
}

__global__ void tankInteractionKernel(int *x,int *y,int *scoring,int *hp, int T,int currentRound,int* hd_ScoreCount,int* hd_HitCount){
    int target_id = (blockIdx.x + currentRound) % T;  // Choose the next tank as the target
    int xc = x[threadIdx.x];
    int yc = y[threadIdx.x];
    __syncthreads();

    // Load tank coordinates and direction into shared memory
    __shared__ int x1, y1, x2, y2, dir;
    if (threadIdx.x == 0) {
        x1 = x[blockIdx.x];
        y1 = y[blockIdx.x];
        x2 = x[target_id];
        y2 = y[target_id];
        // Determine direction
        if (x1 <= x2 && y1 <= y2) dir = 1;
        else if (x1 > x2 && y1 <= y2) dir = 2;
        else if (x1 > x2 && y1 > y2) dir = 3;
        else if (x1 <= x2 && y1 > y2) dir = 4;
    }
    __syncthreads(); 

    bool result = isPointOnLine(xc, yc, x1, y1, x2, y2);
    printf("{%d,%d} & {%d,%d} line is %d collinear to {%d,%d}, dir is %d \n", x1,y1, x2,y2, result, xc,yc, dir);
    __syncthreads();

    __shared__ int it, smallest_dist, old, lockvar;
    if (threadIdx.x == 0) {
        it = -1;
        smallest_dist = 1e9;
        old = 10;
        lockvar = 0;
    }
    
    __syncthreads();  // Ensure all threads have initialized shared memory
    


    do {
    old = atomicCAS(&lockvar, 0,1);
        if(old == 0){
            
            if(hp[blockIdx.x] > 0 && hp[threadIdx.x] > 0){
                int distance = calcDist(x1, y1, xc, yc);
                int direction = calcDir(x1, y1, xc, yc);
                bool lieOnLine = isPointOnLine(xc, yc, x1, y1, x2, y2);
                if (threadIdx.x != blockIdx.x && lieOnLine && direction == dir && distance < smallest_dist)
                {
                    smallest_dist = distance;
                    it = threadIdx.x;
                }
                
            }



            lockvar = 0;
        }
    } while(old!=0);
    __syncthreads();
    if(threadIdx.x == 0){
        printf("%d -> %d and hit to %d\n", blockIdx.x, target_id, it);
        if (it != -1) {
        
          atomicAdd(&hd_HitCount[it],1);
          atomicAdd(&hd_ScoreCount[blockIdx.x],1);
        }
    }
    
    __syncthreads();

}



int main(int argc, char **argv) {
    // Variable declarations
    int M, N, T, H, *xcoord, *ycoord, *score, *hp;

    FILE *inputfilepointer;

    // File Opening for read
    char *inputfilename = argv[1];
    inputfilepointer = fopen(inputfilename, "r");

    if (inputfilepointer == NULL) {
        printf("input.txt file failed to open.");
        return 0;
    }

    fscanf(inputfilepointer, "%d", &M);
    fscanf(inputfilepointer, "%d", &N);
    fscanf(inputfilepointer, "%d", &T); // T is number of Tanks
    fscanf(inputfilepointer, "%d", &H); // H is the starting Health point of each Tank

    // Allocate memory on CPU
    xcoord = (int *)malloc(T * sizeof(int));  // X coordinate of each tank
    ycoord = (int *)malloc(T * sizeof(int));  // Y coordinate of each tank
    score = (int *)malloc(T * sizeof(int));  // Score of each tank (ensure that at the end you have copied back the score calculations on the GPU back to this allocation)
    
    
    // Get the Input of Tank coordinates
    for (int i = 0; i < T; i++) {
        fscanf(inputfilepointer, "%d", &xcoord[i]);
        fscanf(inputfilepointer, "%d", &ycoord[i]);
    }

    auto start = chrono::high_resolution_clock::now();

    //*********************************
    // Your Code begins here (Do not change anything in main() above this comment)
    //********************************

    
    int *x;   // X coordinate of each tank
    int *y;    // Y coordinate of each tank
    int *health;   // Health Points of each tank
    int *scoring; // Score of each tank
    
    hipMalloc(&x, T*sizeof(int));
    hipMalloc(&y, T*sizeof(int));
    hipMalloc(&health, T*sizeof(int));
    hipMalloc(&scoring, T*sizeof(int));
    hp = (int *)malloc(T * sizeof(int));  
    for(int i = 0; i < T; i++){
        hp[i] = H;
    }
    hipMemcpy(x, xcoord, T * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(y, ycoord, T * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(health, hp, T * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(scoring, score, T * sizeof(int), hipMemcpyHostToDevice);
    
    
    
    

    

    int tanksLeft = T;
    int currentRound = 1;
    for(int i = 0; i < T; i++){
      printf("point (%d,%d) score %d Hp %d \n", xcoord[i], ycoord[i], score[i], hp[i]);
    }
    
    while (tanksLeft > 1) {  // Continue rounds until only one or zero tanks left
        if (currentRound % T == 0) {
            currentRound++;
            continue;
        }
        // Allocate memory on CPU
        int *ScoreCount = (int *)malloc(T * sizeof(int));
        int *HitCount = (int *)malloc(T * sizeof(int));
        int *hd_ScoreCount;
        int *hd_HitCount;
        hipMalloc(&hd_ScoreCount, T*sizeof(int));
        hipMalloc(&hd_HitCount, T*sizeof(int));

        hipMemcpy(hd_ScoreCount, ScoreCount, T * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(hd_HitCount, HitCount, T * sizeof(int), hipMemcpyHostToDevice);

        tankInteractionKernel<<<T,T>>>(x,y,scoring,health, T, currentRound, hd_ScoreCount, hd_HitCount);
        hipDeviceSynchronize();
        
        hipMemcpy(ScoreCount, hd_ScoreCount, T * sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(HitCount, hd_HitCount, T * sizeof(int), hipMemcpyDeviceToHost);

        for(int i = 0; i < T; ++i) {
            hp[i] -= HitCount[i];
            score[i] += ScoreCount[i];
            if (hp[i] <= 0) tanksLeft--;
        }
        for(int i = 0; i < T; i++){
            printf("point (%d,%d) score %d Hp %d \n", xcoord[i], ycoord[i], score[i], hp[i]);
        }
        printf("Round %d going to ended\n\n", currentRound);

        currentRound++;
        hipFree(hd_ScoreCount);
        hipFree(hd_HitCount);
        free(ScoreCount);
        free(HitCount);
        
        

    }
    printf("\nchecking code execuete properly or not\n");


    hipMemcpy(hp, health, T * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(score, scoring, T * sizeof(int), hipMemcpyDeviceToHost);

    //*********************************
    // Your Code ends here (Do not change anything in main() below this comment)
    //********************************

    auto end = chrono::high_resolution_clock::now();

    chrono::duration<double, std::micro> timeTaken = end - start;

    printf("Execution time : %f\n", timeTaken.count());

    // Output
    char *outputfilename = argv[2];
    char *exectimefilename = argv[3];
    FILE *outputfilepointer;
    outputfilepointer = fopen(outputfilename, "w");

    for (int i = 0; i < T; i++) {
        fprintf(outputfilepointer, "%d\n", score[i]);
    }
    fclose(inputfilepointer);
    fclose(outputfilepointer);

    outputfilepointer = fopen(exectimefilename, "w");
    fprintf(outputfilepointer, "%f", timeTaken.count());
    fclose(outputfilepointer);

    free(xcoord);
    free(ycoord);
    free(score);
    free(hp);
    
    hipFree(x);
    hipFree(y);
    hipFree(health);
    hipFree(scoring);
    hipDeviceSynchronize();
    return 0;
}
